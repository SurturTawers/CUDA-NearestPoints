
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <algorithm>
#include <chrono>
using namespace std;
struct p3D{
    int pt_num;
    float x;
    float y;
    float z;
};
p3D* crearP3D(float* coords){
    p3D* point=new p3D;
    point->pt_num=coords[0];
    point->x=coords[1];
    point->y=coords[2];
    point->z=coords[3];
    return point;
}
int getInput(string input, float* params){
    int inputSize=input.length(), i=0, k=0;
    string aux;
    while(i<inputSize){ //Guardo uno a uno los parametros en el string aux
        while(input[i]!=' ' && i<inputSize){ // mientras no se encuentre un espacio y no sobrepase el tamaño del input
            aux.push_back(input[i]);
            i++;
        }
        if(aux.size()!=0){ //si hay algún parametro
            params[k]= stof(aux); //lo transformo de string a float y lo guardo en params
            aux.clear();
            k++;
        }
        i++;
    }
    if(k!=4){   //verifico que hayan 4 parámetros
        cout<<"Numero incorrecto de parametros: "<<k<<" de 4"<<endl;
        return 0;
    }
    for(int z=0;z<4;z++){ //verifico que sean valores positivos
        if(params[z]<=0){
            cout<<"Ingrese valores positivos y mayores a cero -> "<<params[z]<<endl;
            return 0;
        }
    }
    if(params[3]>params[0]){
        cout<<"[T] debe ser menor o igual a [N]: "<<params[3]<<" > "<<params[0]<<endl;
        return 0;
    }
    if(params[1]>params[0]){
        cout<<"[K] debe ser menor o igual a [N]: "<<params[1]<<" > "<<params[0]<<endl;
        return 0;
    }
    if(params[0]-floor(params[0])){
        cout<<"Ingrese un numero entero por favor -> "<<params[0]<<endl;
        return 0;
    }
    if(params[1]-floor(params[1])){
        cout<<"Ingrese un numero entero por favor -> "<<params[1]<<endl;
        return 0;
    }
    if(params[3]-floor(params[3])){
        cout<<"Ingrese un numero entero por favor -> "<<params[3]<<endl;
    }
    return 1;
}
void getPoints(string* points, p3D** puntos, int N){
    float coords[4];
    int tam,k=0,j=0;
    string aux;
    for(int i=0;i<N;i++){//i<748720
        tam=points[i].length();
        while(k<tam){//Obtengo los puntos y las coordenadas en el string aux
            while(points[i][k]!=' ' && k<tam){
                aux.push_back(points[i][k]);
                k++;
            }
            if(aux.size()!=0){
                coords[j]=stof(aux);
                aux.clear();
                j++;
            }
            k++;
        }
        puntos[i]=crearP3D(coords);
        k=0;
        j=0;
    }
}
void printPoints(p3D** points, int N){
    cout<<"\n------------PUNTOS------------"<<endl;
    cout<<"pt_num\t\tx\t\t\ty\t\t\tz"<<endl;
    for(int i=0;i<N;i++){//i<748720
        cout<<points[i]->pt_num<<"\t\t"<<points[i]->x<<"\t\t"<<points[i]->y<<"\t\t"<<points[i]->z<<endl;
    }
}
void printSolution(p3D* sol, int K){
    cout<<"\n------------SOLUCION------------"<<endl;
    cout<<"pt_num\t\tx\t\t\ty\t\t\tz"<<endl;
    for(int i=0;i<K;i++){//i<748720
        if(sol[i].pt_num!=0){
            cout<<sol[i].pt_num<<"\t\t"<<sol[i].x<<"\t\t"<<sol[i].y<<"\t\t"<<sol[i].z<<endl;
        }
    }
}
void choosePoints(p3D* chosen, p3D** points, int K, int N){
    int seleccionados[K],num,j=0;
    fill(seleccionados,seleccionados+K,0);
    for(int i=0;i<K;i++){
        num=rand()%N;//obtengo un punto aleatorio
        while(seleccionados[j]!=0 && j<K){
            if(seleccionados[j]==num){//si ya he seleccionado algun punto, seleccion otro
                num=rand()%N;//748720
                j=0;
            }else{
                j++;
            }
        }
        seleccionados[j]=num;
        chosen[i]=*points[num];
    }
}
__global__ void search(p3D* d_points, p3D* d_chosen, p3D* d_solution, int N, int K, float d_m , int T){
    int thid= blockIdx.x * blockDim.x  + threadIdx.x;
    int jmp=blockDim.x*gridDim.x;
    int z=0,k=0;
    float dist,d_x,d_y,d_z;
    __shared__ int cardVec;
    while(k<K){
        for(int j=thid;j<N;j+=jmp){
            d_x= powf(d_chosen[k].x - d_points[j].x,2.0);
            d_y= powf(d_chosen[k].y - d_points[j].y,2.0);
            d_z= powf(d_chosen[k].z - d_points[j].z,2.0);
            dist= sqrtf(d_x + d_y + d_z);
            if(dist<=d_m){
                atomicAdd(&cardVec,1);
            }
        }
        __syncthreads();
        if(cardVec>=T){//si la cardinalidad del punto actual de chosen es mayor a T
            d_solution[z]=d_chosen[k];//lo agrego al arreglo de soluciones
            z++;
        }
        k++;
        cardVec=0;
        __syncthreads();
    }
    __syncthreads();
}
void searchPoints(p3D** points, int N,int K, float d_m, int T){
    int blocks=10,threads=1020;
    p3D* pts=new p3D[N];
    for(int i=0;i<N;i++){
        pts[i]= *points[i];
    }
    p3D* chosen=new p3D[K];
    choosePoints(chosen,points,K,N);//elige los K puntos aleatoriamente
    //seleccioanr si mostrar los puntos elegidos
    string input;
    int option;
    cout<<"\nMostrar puntos elegidos?\n[1]: Si\t[2]: No"<<endl;
    getline(cin,input);
    option=stoi(input);
    while(option!=1 && option!=2){
        cout<<"\nIngrese una opcion valida\nMostrar puntos elegidos?\n[1]: Si\t[2]: No"<<endl;
        getline(cin,input);
        option=stoi(input);
    }
    if(option==1){
        cout<<"\n------------PUNTOS SELECCIONADOS------------"<<endl;
        cout<<"pt_num\t\tx\t\t\ty\t\t\tz"<<endl;
        for(int i=0;i<K;i++){
            cout<<chosen[i].pt_num<<"\t\t"<<chosen[i].x<<"\t\t"<<chosen[i].y<<"\t\t"<<chosen[i].z<<endl;
        }
    }
    p3D* sol=new p3D[K];
    p3D* d_chosen;
    p3D* d_pts;
    p3D* d_sol;
    hipMalloc(&d_sol,K*sizeof(p3D));
    hipMalloc(&d_pts,N*sizeof(p3D));
    hipMalloc(&d_chosen,K*sizeof(p3D));
    hipMemcpy(d_pts,pts,sizeof(p3D)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_chosen,chosen,sizeof(p3D)*K,hipMemcpyHostToDevice);
    double total=0;
    auto start=chrono::high_resolution_clock::now();
    auto end=chrono::high_resolution_clock::now();
    auto time=chrono::duration_cast<chrono::nanoseconds>(end-start).count();
    for(int i=0;i<50;i++){
        start=chrono::high_resolution_clock::now();
        search<<<blocks,threads>>>(d_pts,d_chosen,d_sol,N,K,d_m,T);
        end=chrono::high_resolution_clock::now();
        time=chrono::duration_cast<chrono::nanoseconds>(end-start).count();
        total+=time;
    }
    hipMemcpy(sol,d_sol,K*sizeof(p3D),hipMemcpyDeviceToHost);
    hipFree(d_pts);
    hipFree(d_chosen);
    hipFree(d_sol);
    printSolution(sol,K);
    cout<<"\nTiempo para 50 ejecuciones con "<<blocks<<" bloques y "<<threads<<" hebras: "<<total<<" ns\nPromedio: "<<total/(float)50<<" ns"<<endl;
    delete[] pts;
    delete[] sol;
    delete[] chosen;
}
int main() {
    ifstream puntos;
    puntos.open("/tmp/tmp.iC2UmBBVO8/puntos3D.txt"); //Cambiar por la ruta en la que se encuentra el archivo puntos3D.txt
    if(puntos.is_open()){
        string input;
        float params[4];
        cout<<"Ingrese los valores separados por un espacio\n[N]:\ttamaño del conjunto de puntos (maximo 748742)\n[K]:\tnumero de puntos a analizar.\n[d_m]:\tdistancia máxima entre puntos.\n[T]:\tcardinalidad mínima de la vecindad."<<endl;
        getline(cin,input);
        if(getInput(input,params)){
            int N=params[0],K=params[1];
            srand(time(NULL));
            string* point=new string[N];
            p3D** points=new p3D*[N];
            for(int i=0;i<N;i++){
                getline(puntos,point[i]);
            }
            getPoints(point,points,N);
            string input;
            int option;
            cout<<"\nMostrar puntos?\n[1]: Si\t[2]: No"<<endl;
            getline(cin,input);
            option=stoi(input);
            while(option!=1 && option!=2){
                cout<<"\nIngrese una opcion valida\nMostrar puntos?\n[1]: Si\t[2]: No"<<endl;
                getline(cin,input);
                option=stoi(input);
            }
            if(option==1){
                printPoints(points, N);
            }
            searchPoints(points,N,K,params[2],params[3]);
            cout<<"\nAdios :)"<<endl;
            for(int i=0;i<N;i++){
                delete[] points[i];
            }
            delete[] point;
        }else{
            cout<<"Adios :("<<endl;
            puntos.close();
            return 0;
        }
    }else{
        cout<<"No se pudo leer el archivo con los puntos :( adios"<<endl;
    }
    puntos.close();
    return 0;
}